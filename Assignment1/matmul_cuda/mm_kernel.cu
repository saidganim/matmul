#include "hip/hip_runtime.h"
extern "C"
{
#include "mm_kernel.h"
}
#include <hip/hip_runtime.h>


__global__ void matrix_kernel(int m, int n, int p, float* A, float* B, float* C){

	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned j = i % p;
	i = i / p;
	
	if(i >= m || j >= p)
		return;
	
	for(int k=0; k<n; k++) {
	        C[i*p+j] += A[i*n+k]*B[k*p+j];
	}

}


extern "C"
void matrix_mult(int m, int n, int p, float *A, float *B, float *C) {
  //int i, j, k;
	int threadBlock = 512;
	float *dA, *dB, *dC;
	hipMalloc(&dA, m * n * sizeof(float));
	hipMalloc(&dB, p * n * sizeof(float));
	hipMalloc(&dC, m * p * sizeof(float));
	
	//if(hipGetLastError != hipSuccess){
	//	printf("CUDA_ERROR\n");
	//	exit(1);
	//}

	hipMemcpy(dA, A, m*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, p*n*sizeof(float), hipMemcpyHostToDevice);

	matrix_kernel<<<m * n / threadBlock + 1, threadBlock>>>(m,n,p,dA,dB,dC);
	hipMemcpy(C, dC, m*p*sizeof(float), hipMemcpyDeviceToHost);

//  for(i=0; i<m; i++) {
  //  for(j=0; j<p; j++) {
    //  C[i*p+j]=0;
     // for(k=0; k<n; k++) {
       // C[i*p+j] += A[i*n+k]*B[k*p+j];
//      }
  //  }
//  }
}
