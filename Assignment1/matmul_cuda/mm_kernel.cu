#include "hip/hip_runtime.h"


#include "mm_kernel.h"

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

__global__ void matrix_kernel(int m, int n, int p, float* A, float* B, float* C){

	unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned j = i % p;
	i = i / p;

	if(i >= m)
		return;

	for(int k=0; k<n; k++) {
	        C[i*p+j] += A[i*n+k]*B[k*p+j];
	}

}


void matrix_mult(int m, int n, int p, float *A, float *B, float *C) {
  //int i, j, k;
	struct timeval start, end;
	int threadBlock = 512;
	float *dA, *dB, *dC;
	hipMalloc(&dA, m * n * sizeof(float));
	hipMalloc(&dB, p * n * sizeof(float));
	hipMalloc(&dC, m * p * sizeof(float));

	//if(hipGetLastError != hipSuccess){
	//	printf("CUDA_ERROR\n");
	//	exit(1);
	//}

	hipMemcpy(dA, A, m*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dB, B, p*n*sizeof(float), hipMemcpyHostToDevice);
	gettimeofday(&start, 0);
	matrix_kernel<<<m * p / threadBlock + 1, threadBlock>>>(m,n,p,dA,dB,dC);
	hipDeviceSynchronize();
	gettimeofday(&end, 0);
	printf("time without memory copy = %f\n", end.tv_sec + end.tv_usec/1000000.0 - (start.tv_sec + start.tv_usec / 1000000.0));
	hipMemcpy(C, dC, m*p*sizeof(float), hipMemcpyDeviceToHost);

//  for(i=0; i<m; i++) {
  //  for(j=0; j<p; j++) {
    //  C[i*p+j]=0;
     // for(k=0; k<n; k++) {
       // C[i*p+j] += A[i*n+k]*B[k*p+j];
//      }
  //  }
//  }
}
